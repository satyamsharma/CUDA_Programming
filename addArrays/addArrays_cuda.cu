
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h> // provides resolution of 1 us

//Number of threads in one thread block
#define THREAD_NUM (256) 

// cuda kernel to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for(int i = 0; i < n; i++)
		y[i] = x[i] + y[i];
}


int main(void)
{
	timeval t1, t2;
	double elapsedTime;

	int N = 1 << 20;

	float *x, *y;

	//Allocate Unified memory -accessible from both CPU or GPU 🤘
	//Conceptually, foes the same function as new 
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for(int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	} 

	// timestamp t1
	gettimeofday(&t1, NULL);

	//Run kernel on 1M elements on the CPU
	add<<<1, THREAD_NUM>>>(N, x, y);

	// timestamp t2
    gettimeofday(&t2, NULL);

	//Wait for GPU to finish before accessing on host
	//Why: to make CPU wait from accessing GPU result
	hipDeviceSynchronize();

	//Free memory
	hipFree(x);
	hipFree(y);


	// compute and print the elapsed time in millisec
    elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;   // us to ms

	std::cout << "Amount of time to add 1 Million elements (with CUDA cores): " << elapsedTime << " millisec." << std::endl;


	return 0;
}